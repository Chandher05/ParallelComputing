#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#define inf 9999
#define N 1000





int main(int argc, char **argv) {

    struct timeval first, second, lapsed, third;
    struct timezone tzp, tzp2;
    float *host_A;
    int *host_Q;
    float *dev_x;
    int *dev_qx;
    float *A;
    int *Q;
    float *D;
    

    int i, j, bk;
    int k = 0;
    //int n = atoi(argv[1]);
    int n = N;

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    printf("\n");
    printf("RUNNING WITH %d VERTICES \n", n);
    printf("\n");

    hipMalloc(&dev_x, n * n * sizeof (float));
    hipMalloc(&dev_qx, n * n * sizeof (float));

    //CPU arrays
    A = (float *) malloc(n * n * sizeof (float)); //arxikos pinakas A
    D = (float *) malloc(n * n * sizeof (float)); //arxikos pinakas D
    Q = (int *) malloc(n * n * sizeof (int)); //arxikos pinakas Q

    //GPU arrays
    host_A = (float *) malloc(n * n * sizeof (float));
    host_Q = (int *) malloc(n * n * sizeof (int));

    srand(time(NULL));
    
    gettimeofday(&third, &tzp2);
    ////////////////////////////First Mem Copy////////////////////
    gettimeofday(&first, &tzp);
    hipMemcpy(dev_x, host_A, n * n * sizeof (float), hipMemcpyHostToDevice);
    hipMemcpy(dev_qx, host_Q, n * n * sizeof (int), hipMemcpyHostToDevice);
    gettimeofday(&second, &tzp);
    if (first.tv_usec > second.tv_usec) {
        second.tv_usec += 1000000;
        second.tv_sec--;
    }
    lapsed.tv_usec = second.tv_usec - first.tv_usec;
    lapsed.tv_sec = second.tv_sec - first.tv_sec;
    printf("First Transfer CPU to GPU  Time elapsed: %lu, %lu s\n", lapsed.tv_sec, lapsed.tv_usec);
    ////////////////////////////////////////////////////GPU Calculation////////////////////////////////

    bk = (int) (n * n / 512);
    int gputhreads = 512;
    if (bk > 0) {
        gputhreads = 512;
    } else {
        bk = 1;
        gputhreads = n*n;
    }
    printf(" \n");
    printf("BLOCKS :   %d      GPU THREADS:     %d \n", bk, gputhreads);
    printf(" \n");
    //gettimeofday(&first, &tzp);
    hipEventRecord(start); 
    funct << <bk, gputhreads>>>(n, k, dev_x, dev_qx);
    hipDeviceSynchronize();
    //gettimeofday(&second, &tzp);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    if (first.tv_usec > second.tv_usec) {
        second.tv_usec += 1000000;
        second.tv_sec--;
    }

    lapsed.tv_usec = second.tv_usec - first.tv_usec;
    lapsed.tv_sec = second.tv_sec - first.tv_sec;
    printf("GPU Calculation Time elapsed: %.20f\n", milliseconds * .0001);
    printf("\n");

    
    printf("ALL OK WE ARE DONE \n");
    return 0;
}