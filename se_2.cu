#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
long datasize;

hipError_t searchKeyword(int *result, char *data, char *keyword);
__global__ void searchKeywordKernel(int *result, char *data, char *keyword,int datasize)
{
 int i = blockIdx.x*threadIdx.x;
 // Detect the first matching character
 if(i<datasize){

 if (data[i] == keyword[0]) {
   // Loop through next keyword character
   for (int j=1; j<3; i++) {
     if (data[i+j] != keyword[j])
       break;
     else
     // Store the first matching character to the result list
       result[i] = 1;
   }
  }
 }
}
int main()
{
    FILE *f = fopen("input.txt", "rb");
    fseek(f, 0, SEEK_END);
    datasize = ftell(f);
    fseek(f, 0, SEEK_SET);  //same as rewind(f);
    fclose(f);

     char data[datasize];
     fread(data, datasize, 1, f);
     // char keyword[8]={'S','H','E','R','L','O','C','K'};// char pattern
     char keyword[8]={'s'};
     int result[datasize];
     // Set false value in result array
     memset(result, 0, datasize);
     // Generate input data

    // Search keyword in parallel.
    hipError_t cudaStatus = searchKeyword(result, data, keyword);
     
    // Print out the string match result position
    int total_matches = 0;
    for (int i=0; i<datasize; i++) {
      if (result[i] == 1) {
        printf("Character found at position % i\n", i);
        total_matches++;
      }
    }
    printf("Total matches = %d\n", total_matches);
    cudaStatus = hipDeviceReset();
    
    system("pause");
    return 0;
     
    }
    // Helper function for using CUDA to search a list of characters in parallel.
hipError_t searchKeyword(int *result, char *data, char *keyword)
{
     char *dev_data = 0;
     char *dev_keyword = 0;
     int *dev_result = 0;
     hipError_t cudaStatus;
     cudaStatus = hipSetDevice(0);
     cudaStatus = hipMalloc((void**)&dev_result, datasize * sizeof(int));
     cudaStatus = hipMalloc((void**)&dev_data, datasize * sizeof(char));
     cudaStatus = hipMalloc((void**)&dev_keyword, datasize * sizeof(char));
     
     cudaStatus = hipMemcpy(dev_data, data, datasize * sizeof(char), hipMemcpyHostToDevice);
     cudaStatus = hipMemcpy(dev_keyword, keyword, datasize * sizeof(char), hipMemcpyHostToDevice);
      
      int bk = (int) (datasize/512);
      int gputhreads = 512;
      if (bk > 0) {
        gputhreads = 512;
      } 
      else{
        bk = 1;
        gputhreads = datasize;
      }
      searchKeywordKernel<<<bk,gputhreads >>>(dev_result, dev_data, dev_keyword,datasize);
      cudaStatus = hipDeviceSynchronize();
      cudaStatus = hipMemcpy(result, dev_result, datasize * sizeof(int), hipMemcpyDeviceToHost);
      
        hipFree(dev_result);
        hipFree(dev_data);
        hipFree(dev_keyword);
     
     return cudaStatus;
     }